
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

/**
Max size 1024
*/
__global__ void kreduce(unsigned int *vec, int size){
	int tid = threadIdx.x;
	for(int offset=(size/2);offset >= 1;offset /= 2){
		if(tid < offset){
			vec[tid] += vec[tid+offset];
		}
		__syncthreads();
	}

}

void reduce(unsigned int *vec, unsigned int *sum, int size){
	unsigned int *d_vec;
	int bytes = size*sizeof(unsigned int);

	hipMalloc((void **)&d_vec, bytes);
	hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);

	kreduce<<<1, size>>>(d_vec, size);

	hipMemcpy(sum, d_vec, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(d_vec);
}

int main(int argc, char **argv)
{
  if (argc < 2){
    printf("Usage: <filename>\n");
    exit(-1);
  }
  int size;
  unsigned int *vec;
  FILE *f = fopen(argv[1],"r");
  fscanf(f,"%d\n",&size);
  if (size >= 20){
    printf("Size (%u) is too large: size is limited to 2^10\n",size);
    exit(-1);
  }
  size = 1 << size;
   vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);
  for (int i=0; i<size; i++){
    fscanf(f, "%u\n",&(vec[i]));
  }

  unsigned int sum;

	reduce(vec,&sum,size);

	printf("%u\n", sum);
}
