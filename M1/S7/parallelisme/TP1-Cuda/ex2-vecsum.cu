
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

/**
Max size 1024
*/
__global__ void kreduce(unsigned int *vec, int size){
  int tid = threadIdx.x;
  int gid = blockIdx.x * blockDim.x + tid;

	for(int offset=(size/2);offset >= 1;offset /= 2){
		if(tid < offset){
			vec[gid] += vec[gid+offset];
		}
		__syncthreads();
  }
  if (threadIdx.x == 0){
    vec[blockIdx.x] = vec[gid];
  }

  __syncthreads();

}

void reduce(unsigned int *vec, unsigned int *sum, int size){
	unsigned int *d_vec;

  unsigned int nbBlocks;

  if (size > 10){
    nbBlocks = 1 << (size - 10);
    size = 1024;
  } else {
    nbBlocks = 1;
    size = 1 << size; 
    printf("%d\n", size);
  }
  
  int bytes = size*sizeof(unsigned int);

	hipMalloc((void **)&d_vec, bytes);
	hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);

  kreduce<<<nbBlocks, size>>>(d_vec, size);

  if (nbBlocks > 1){
    kreduce <<<1, nbBlocks>>>(d_vec, nbBlocks);
  }

	hipMemcpy(sum, d_vec, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(d_vec);
}

/**
Change in how the size is managed : we now pass it as a log2 to the reduce function, which handles the 2^ization
*/
int main(int argc, char **argv)
{
  if (argc < 2){
    printf("Usage: <filename>\n");
    exit(-1);
  }
  int size;
  unsigned int *vec;
  FILE *f = fopen(argv[1],"r");
  fscanf(f,"%d\n",&size);
  if (size >= 20){
    printf("Size (%u) is too large: size is limited to 2^20\n",size);
    exit(-1);
  }

   vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);
  for (int i=0; i<size; i++){
    fscanf(f, "%u\n",&(vec[i]));
  }

  unsigned int sum;

	reduce(vec,&sum,size);

	printf("%u\n", sum);
}
